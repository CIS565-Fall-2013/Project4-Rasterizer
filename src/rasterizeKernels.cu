#include "hip/hip_runtime.h"
// CIS565 CUDA Rasterizer: A simple rasterization pipeline for Patrick Cozzi's CIS565: GPU Computing at the University of Pennsylvania
// Written by Yining Karl Li, Copyright (c) 2012 University of Pennsylvania

#include <stdio.h>
#include <hip/hip_runtime.h>
#include <cmath>
#include <thrust/random.h>
#include "rasterizeKernels.h"
#include "rasterizeTools.h"
#include "glm/glm.hpp"
#include "util.h"
#include "variables.h"

#if CUDA_VERSION >= 5000
    #include <hip/hip_vector_types.h>
#else
    #include <cutil_math.h>
#endif

glm::vec3* framebuffer = 0;
fragment* depthbuffer = 0;
float* device_vbo = 0;   //pre-transformed
float* device_vbo_t = 0; //post-transformed
float* device_cbo = 0;
int* device_ibo = 0;
triangle* primitives = 0;

void checkCUDAError(const char *msg) {
  hipError_t err = hipGetLastError();
  if( hipSuccess != err) {
    fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString( err) ); 
    exit(EXIT_FAILURE); 
  }
} 

//Handy dandy little hashing function that provides seeds for random number generation
__host__ __device__ unsigned int hash(unsigned int a){
    a = (a+0x7ed55d16) + (a<<12);
    a = (a^0xc761c23c) ^ (a>>19);
    a = (a+0x165667b1) + (a<<5);
    a = (a+0xd3a2646c) ^ (a<<9);
    a = (a+0xfd7046c5) + (a<<3);
    a = (a^0xb55a4f09) ^ (a>>16);
    return a;
}

//Writes a given fragment to a fragment buffer at a given location
__host__ __device__ void writeToDepthbuffer(int x, int y, fragment frag, fragment* depthbuffer, ushort width, ushort height){
  if( x < width && y < height){

    depthbuffer[(y*width) + x] = frag;
  }
}

//Reads a fragment from a given location in a fragment buffer
__host__ __device__ fragment getFromDepthbuffer(int x, int y, fragment* depthbuffer, ushort width, ushort height){
  if(x<width && y<height)
  {

    return depthbuffer[(y*width) + x];
  }
  else
  {
    fragment f;
    return f;
  }
}

//Writes a given pixel to a pixel buffer at a given location
__host__ __device__ void writeToFramebuffer(int x, int y, glm::vec3 value, glm::vec3* framebuffer, ushort width, ushort height){
  if(x<width && y<height){
 
    framebuffer[(y*width) + x] = value;
  }
}

//Reads a pixel from a pixel buffer at a given location
__host__ __device__ glm::vec3 getFromFramebuffer(int x, int y, glm::vec3* framebuffer, ushort width, ushort height){
  if(x<width && y<height){

    return framebuffer[(y*width) + x];
  }else{
    return glm::vec3(0,0,0);
  }
}

//Kernel that clears a given pixel buffer with a given color
__global__ void clearImage( ushort width, ushort height, glm::vec3* image, glm::vec3 color){
    int x = (blockIdx.x * blockDim.x) + threadIdx.x;
    int y = (blockIdx.y * blockDim.y) + threadIdx.y;
    int index = x + (y * width);
    if(x<width && y<height)
    {
      image[index] = color;
    }
}

//Kernel that clears a given fragment buffer with a given fragment
__global__ void clearDepthBuffer( ushort width, ushort height, fragment* buffer, fragment frag){
    int x = (blockIdx.x * blockDim.x) + threadIdx.x;
    int y = (blockIdx.y * blockDim.y) + threadIdx.y;
    int index = x + (y * width);
    if(x<width && y<height)
    {
      fragment f = frag;
      f.position.x = x;
      f.position.y = y;
      buffer[index] = f;
    }
}

//Kernel that writes the image to the OpenGL PBO directly. 
__global__ void sendImageToPBO(uchar4* PBOpos, ushort width, ushort height, glm::vec3* image){
  
  int x = (blockIdx.x * blockDim.x) + threadIdx.x;
  int y = (blockIdx.y * blockDim.y) + threadIdx.y;
  int index = x + (y * width);
  
  if(x<width && y<height){

      glm::vec3 color;      
      color.x = image[index].x*255.0;
      color.y = image[index].y*255.0;
      color.z = image[index].z*255.0;

      if(color.x>255){
        color.x = 255;
      }

      if(color.y>255){
        color.y = 255;
      }

      if(color.z>255){
        color.z = 255;
      }
      
      // Each thread writes one pixel location in the texture (textel)
      PBOpos[index].w = 0;
      PBOpos[index].x = color.x;     
      PBOpos[index].y = color.y;
      PBOpos[index].z = color.z;
  }
}

//TODO: Implement a vertex shader
__global__ void vertexShadeKernel(float* vbo, float* vbo_t, ushort width, ushort height, int vbosize, VertUniform uniform)
{
  int index = (blockIdx.x * blockDim.x) + threadIdx.x;
  glm::vec4 pos;
  if( index < vbosize/3 )
  {
      pos.x = vbo[ 3*index ];
      pos.y = vbo[ 3*index+1 ];
      pos.z = vbo[ 3*index+2 ];
      pos.w = 1.0f;

      pos = uniform.viewingMat * pos;
      pos = uniform.projMat * pos;

      //Perspective divide
      pos.x /= pos.w;
      pos.y /= pos.w;
      pos.z /= pos.w;

      //convert to window coordinate
      pos.x = width * ( pos.x + 1.0f ) /2.0f;
      pos.y = height * ( pos.y + 1.0f ) / 2.0f;
      //memcpy( &vbo[3*index], &pos[0], sizeof(float)*3 );
      vbo_t[3*index] = pos.x;
      vbo_t[3*index+1] = pos.y;
      vbo_t[3*index+2] = pos.z;
  }
}

//TODO: Implement primative assembly
__global__ void primitiveAssemblyKernel(float* vbo, int vbosize, float* cbo, int cbosize, int* ibo, int ibosize, triangle* primitives)
{
  int index = (blockIdx.x * blockDim.x) + threadIdx.x;
  int primitivesCount = ibosize/3;

  triangle pri;
  size_t size;
  glm::vec3 orient;
  if(index<primitivesCount)
  {
      size = sizeof(float)*3;
      memcpy( &pri.p[0][0], &vbo[3*ibo[3*(index)]   ], size );
      memcpy( &pri.p[1][0], &vbo[3*ibo[3*(index)+1] ], size );
      memcpy( &pri.p[2][0], &vbo[3*ibo[3*(index)+2] ], size );
      primitives[index] = pri;
  }
}

//TODO: Implement a rasterization method, such as scanline.
__global__ void rasterizationKernel(triangle* primitives, int primitivesCount, fragment* depthbuffer, ushort width, ushort height)
{
  int index = (blockIdx.x * blockDim.x) + threadIdx.x;

  triangle pri;
  int y1, y2, y3, x1, x2, x3;

  int Dx12, Dx23, Dx31;
  int Dy12, Dy23, Dy31;
  int minx, maxx, miny, maxy;
  int C1, C2, C3;
  float Cy1, Cy2, Cy3;
  int Cx1, Cx2, Cx3;

  char base1, base2, base3;
  float zval;
  float x_alpha, z_alpha;
  float x_l, x_r, z_l, z_r, w_l, w_r;
  glm::vec2 t_l, t_r;
  glm::vec3 color_l, color_r;
  x_l = 0;
  if(index<primitivesCount)
  {
      pri = primitives[index];
      y1 = pri.p[0].y + 0.5f;
      x1 = pri.p[0].x + 0.5f;

      y2 = pri.p[1].y + 0.5f;
      x2 = pri.p[1].x + 0.5f;

      y3 = pri.p[2].y + 0.5f;
      x3 = pri.p[2].x + 0.5f;

      //delta
      Dx12 = x2 - x1;
      Dx23 = x3 - x2;
      Dx31 = x1 - x3;

      Dy12 = y2 - y1;
      Dy23 = y3 - y2;
      Dy31 = y1 - y3;

      //Bounding coordinate
      minx = min( min( x1, x2 ),x3 );
      maxx = max( max( x1, x2 ), x3 );
      miny = min( min( y1, y2 ), y3 );
      maxy = max( max( y1, y2 ), y3 );
      if( minx < 0 ) minx = 0;
      if( miny < 0 ) miny = 0;
      if( maxx >= width ) maxx = width;
      if( maxy >= height ) maxy = height;

      //constant part of half-edge functions
      
      C1 = Dy12*x1 - Dx12*y1; //derived from line equation (X1-X2)*(y-Y1) - (Y1-Y2)*(x-X1) = 0
      C2 = Dy23*x2 - Dx23*y2; //derived from line equation (X2-X3)*(y-Y2) - (Y2-Y3)*(x-X2) = 0
      C3 = Dy31*x3 - Dx31*y3; //derived from line equation (X3-X1)*(y-Y3) - (Y3-Y1)*(x-X3) = 0

      //Correct for fill convention
      if( Dy12 < 0 || ( Dy12 == 0 && Dx12 > 0 ) )
          C1 += 1;
      if( Dy23 < 0 || ( Dy23 == 0 && Dx23 > 0 ) )
          C2 += 1;
      if( Dy31 < 0 || ( Dy31 == 0 && Dx31 > 0 ) )
          C3 += 1;

      Cy1 = C1 + Dx12 * miny - Dy12 * minx;
      Cy2 = C2 + Dx23 * miny - Dy23 * minx;
      Cy3 = C3 + Dx31 * miny - Dy31 * minx;

      for( int y = miny; y < maxy; ++y )
      {
          Cx1 = Cy1;
          Cx2 = Cy2;
          Cx3 = Cy3;
          for( int x = minx; x < maxx; ++x )
          {
              if( Cx1 > 0 && Cx2 > 0 && Cx3 > 0 )
              {
                  //interpolate attributes using barycentic interpolation
               //   if( pri.p[0].y - pri.p[1].y != 0 && pri.p[0].y - pri.p[2].y != 0 )
               //   {
               //       base1 = 0; base2 = 1; base3 = 2;
               //   }
               //   else if( pri.p[1].y - pri.p[0].y != 0 && pri.p[1].y - pri.p[2].y != 0 )
               //   {
               //       base1 = 1; base2 = 0; base3 = 2;
               //   }
               //   else
               //   {
               //       base1 = 2; base2 = 0; base3 =1;
               //   }

               //   //interpolate Z value
               //   x_alpha = ( pri.p[base1].x - pri.p[base2].x ) / ( pri.p[base1].y - pri.p[base2].y );
               //   x_l = ( y - pri.p[base2].y ) * x_alpha + pri.p[base2].x;

               //   x_alpha = ( pri.p[base1].x - pri.p[base3].x ) / ( pri.p[base1].y - pri.p[base3].y );
               //   x_r = ( y - pri.p[base3].y ) * x_alpha + pri.p[base3].x;

               //   z_alpha = ( pri.p[base1].z - pri.p[base2].z ) / ( pri.p[base1].y - pri.p[base2].y );
               //   z_l = ( y - pri.p[base2].y ) * z_alpha + pri.p[base2].z;

               //   z_alpha = ( pri.p[base1].z - pri.p[base3].z ) / ( pri.p[base1].y - pri.p[base3].y );
               //   z_r = ( y - pri.p[base3].y ) * z_alpha + pri.p[base3].z;

               //   zval = ( ( z_l - z_r ) / (float)( x_l - x_r )) * ( x - x_r ) + z_r; 
               //   //if( zval > depthbuffer[y*width + x]. )
               //   //    continue;

               //   x_alpha = ( y - pri.p[base2].y ) / ( pri.p[base2].y - pri.p[base1].y );
               //   x_l = pri.p[base2].x + x_alpha *( pri.p[base2].x - pri.p[base1].x );
               //   w_l = 1.0f / pri.p[base2].z + x_alpha * ( 1.0f/pri.p[base2].z - 1.0f/pri.p[base1].z );

               //   t_l = pri.t[base2] / pri.p[base2].z + x_alpha * ( pri.t[base2] / pri.p[base2].z- pri.t[base1] / pri.p[base1].z );
               //   color_l = pri.c[base2] / pri.p[base2].z + x_alpha * ( pri.c[base2] / pri.p[base2].z - pri.c[base1] / pri.p[base1].z );

	              //x_alpha = ( y - pri.p[base3].y ) / ( pri.p[base3].y - pri.p[base1].y );
	              //x_r = pri.p[base3].x + x_alpha * ( pri.p[base3].x - pri.p[base1].x );
	              //w_r = 1.0f/pri.p[base3].z + x_alpha * ( 1.0f/pri.p[base3].z - 1.0f/pri.p[base1].z );

	              //t_r = pri.t[base3] / pri.p[base3].z + x_alpha * ( pri.t[base3] / pri.p[base3].z - pri.t[base1] / pri.p[base1].z );
	              //color_r = pri.c[base3] / pri.p[base3].z + x_alpha * ( pri.c[base3] / pri.p[base3].z - pri.c[base1] /pri.p[base1].z );

               //   x_alpha = ( x-x_r )/(x_l-x_r);
                  //unsigned int assume = __float_as_int(depthbuffer[x+(width*y)].color.x);
                  //atomicCAS( (int*)&depthbuffer[x+(width*y)].color.x, assume, 1 );
                  depthbuffer[x+(width*y)].color.x = 1.0f;
                  depthbuffer[x+(width*y)].color.y = 1.0f;
                  depthbuffer[x+(width*y)].color.z = 1.0f;
              } 
              Cx1 -= Dy12;
              Cx2 -= Dy23;
              Cx3 -= Dy31;
          }
          Cy1 += Dx12;
          Cy2 += Dx23;
          Cy3 += Dx31;
      }
  }
}

//TODO: Implement a fragment shader
__global__ void fragmentShadeKernel(fragment* depthbuffer, ushort width, ushort height){
  int x = (blockIdx.x * blockDim.x) + threadIdx.x;
  int y = (blockIdx.y * blockDim.y) + threadIdx.y;
  int index = x + (y * width);
  if(x<width && y<height){
     
  }
}

//Writes fragment colors to the framebuffer
__global__ void render(ushort width, ushort height, fragment* depthbuffer, glm::vec3* framebuffer){

  int x = (blockIdx.x * blockDim.x) + threadIdx.x;
  int y = (blockIdx.y * blockDim.y) + threadIdx.y;
  int index = x + (y * width);

  if(x<width && y<height){
    framebuffer[index] = depthbuffer[index].color;
  }
}

// Wrapper for the __global__ call that sets up the kernel calls and does a ton of memory management
void cudaRasterizeCore(uchar4* PBOpos, ushort width, ushort height, float frame, float* vbo, int vbosize, float* cbo, int cbosize, int* ibo, int ibosize,
                       VertUniform &vsUniform, FragUniform &fsUniform)
{

  // set up crucial magic
  int tileSize = 8;
  dim3 blockSize( tileSize, tileSize );
  dim3 gridSize(( width + blockSize.x -1)/blockSize.x, (height+blockSize.y-1)/blockSize.y );  

  //set up framebuffer
  //framebuffer = NULL;
  //cudaErrorCheck( hipMalloc((void**)&framebuffer, width*height*sizeof(glm::vec3)) );
  //
  ////set up depthbuffer
  //depthbuffer = NULL;
  //cudaErrorCheck( hipMalloc((void**)&depthbuffer, width*height*sizeof(fragment)) );

  //kernel launches to black out accumulated/unaccumlated pixel buffers and clear our scattering states
  clearImage<<<gridSize, blockSize>>>(width,height, framebuffer, glm::vec3(0,0,0));
  
  fragment frag;
  frag.color = glm::vec3(0,0,0);
  frag.normal = glm::vec3(0,0,0);
  frag.position = glm::vec3(0,0,-10000);
  clearDepthBuffer<<<gridSize, blockSize>>>(width,height, depthbuffer,frag);

  //------------------------------
  //memory stuff
  //------------------------------
  //primitives = NULL;
  //cudaErrorCheck(  hipMalloc((void**)&primitives, (ibosize/3)*sizeof(triangle)) );

  //device_ibo = NULL;
  //cudaErrorCheck(  hipMalloc((void**)&device_ibo, ibosize*sizeof(int)) );
  //cudaErrorCheck(  hipMemcpy( device_ibo, ibo, ibosize*sizeof(int), hipMemcpyHostToDevice) );

  //device_vbo = NULL;
  //cudaErrorCheck( hipMalloc((void**)&device_vbo, vbosize*sizeof(float)) );
  //cudaErrorCheck( hipMemcpy( device_vbo, vbo, vbosize*sizeof(float), hipMemcpyHostToDevice) );

  //device_cbo = NULL;
  //cudaErrorCheck( hipMalloc((void**)&device_cbo, cbosize*sizeof(float)) );
  //cudaErrorCheck( hipMemcpy( device_cbo, cbo, cbosize*sizeof(float), hipMemcpyHostToDevice) );
 
  tileSize = 32;
  int primitiveBlocks = (vbosize/3 + tileSize-1)/tileSize;
  //------------------------------
  //vertex shader
  //------------------------------
  vertexShadeKernel<<<primitiveBlocks, tileSize>>>(device_vbo, device_vbo_t, width, height, vbosize, vsUniform );

  cudaErrorCheck(  hipDeviceSynchronize() );
  //------------------------------
  //primitive assembly
  //------------------------------
  primitiveBlocks = (ibosize/3 + tileSize-1 )/tileSize;
  primitiveAssemblyKernel<<<primitiveBlocks, tileSize>>>(device_vbo_t, vbosize, device_cbo, cbosize, device_ibo, ibosize, primitives);

  cudaErrorCheck( hipDeviceSynchronize() );
  //------------------------------
  //rasterization
  //------------------------------
  rasterizationKernel<<<primitiveBlocks, tileSize>>>(primitives, ibosize/3, depthbuffer, width, height);

  cudaErrorCheck( hipDeviceSynchronize() );
  //------------------------------
  //fragment shader
  //------------------------------
  fragmentShadeKernel<<<gridSize, blockSize>>>(depthbuffer, width, height);

  cudaErrorCheck( hipDeviceSynchronize() );
  //------------------------------
  //write fragments to framebuffer
  //------------------------------
  render<<<gridSize, blockSize>>>(width, height, depthbuffer, framebuffer);
  sendImageToPBO<<<gridSize, blockSize>>>(PBOpos, width, height, framebuffer);

  cudaErrorCheck( hipDeviceSynchronize() );

  //kernelCleanup();

  checkCUDAError("Kernel failed!");
}

void initDeviceBuf( ushort width, ushort height, float* vbo, int vbosize, float* cbo, int cbosize, 
                 int* ibo, int ibosize )
{
    kernelCleanup();

    //create framebuffer
    framebuffer = 0;
    cudaErrorCheck( hipMalloc((void**)&framebuffer, width*height*sizeof(glm::vec3)) );

    //create depth buffer
    depthbuffer = NULL;
    cudaErrorCheck( hipMalloc((void**)&depthbuffer, width*height*sizeof(fragment)) );

    //------------------------------
    //memory stuff
    //------------------------------
    primitives = NULL;
    cudaErrorCheck(  hipMalloc((void**)&primitives, (ibosize/3)*sizeof(triangle)) );

    device_ibo = NULL;
    cudaErrorCheck(  hipMalloc((void**)&device_ibo, ibosize*sizeof(int)) );
    cudaErrorCheck(  hipMemcpy( device_ibo, ibo, ibosize*sizeof(int), hipMemcpyHostToDevice) );

    device_vbo = NULL;
    cudaErrorCheck( hipMalloc((void**)&device_vbo, vbosize*sizeof(float)) );
    cudaErrorCheck( hipMemcpy( device_vbo, vbo, vbosize*sizeof(float), hipMemcpyHostToDevice) );

    device_vbo_t = NULL;
    cudaErrorCheck( hipMalloc((void**)&device_vbo_t, vbosize*sizeof(float)) );

    device_cbo = NULL;
    cudaErrorCheck( hipMalloc((void**)&device_cbo, cbosize*sizeof(float)) );
    cudaErrorCheck( hipMemcpy( device_cbo, cbo, cbosize*sizeof(float), hipMemcpyHostToDevice) );
}

void kernelCleanup()
{
  if( primitives )
  {
      hipFree( primitives );
      primitives = 0;
  }
  if( device_vbo )
  {
      hipFree( device_vbo );
      device_vbo = 0;
  }
  if( device_vbo_t )
  {
      hipFree( device_vbo_t );
      device_vbo_t = 0;
  }
  if( device_cbo )
  {
      hipFree( device_cbo );
      device_cbo = 0;
  }
  if( device_ibo )
  {
      hipFree( device_ibo );
      device_ibo = 0;
  }
  if( framebuffer )
  {
      hipFree( framebuffer );
      framebuffer = 0;
  }
  if( depthbuffer )
  {
      hipFree( depthbuffer );
      depthbuffer = 0;
  }

}

