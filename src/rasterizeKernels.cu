#include "hip/hip_runtime.h"
// CIS565 CUDA Rasterizer: A simple rasterization pipeline for Patrick Cozzi's CIS565: GPU Computing at the University of Pennsylvania
// Written by Yining Karl Li, Copyright (c) 2012 University of Pennsylvania

#include <stdio.h>
#include <hip/hip_runtime.h>
#include <cmath>
#include <thrust/random.h>
#include "rasterizeKernels.h"
#include "rasterizeTools.h"
#include "glm/gtc/matrix_transform.hpp"
#define NATHANS_EPSILON 0.0001

#if CUDA_VERSION >= 5000
    #include <hip/hip_vector_types.h>
#else
    #include <cutil_math.h>
#endif

glm::vec3* framebuffer;
fragment* depthbuffer;
float* device_vbo;
float* device_cbo;
int* device_ibo;
triangle* primitives;

void checkCUDAError(const char *msg) {
  hipError_t err = hipGetLastError();
  if( hipSuccess != err) {
    fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString( err) ); 
    exit(EXIT_FAILURE); 
  }
} 

//Handy dandy little hashing function that provides seeds for random number generation
__host__ __device__ unsigned int hash(unsigned int a){
    a = (a+0x7ed55d16) + (a<<12);
    a = (a^0xc761c23c) ^ (a>>19);
    a = (a+0x165667b1) + (a<<5);
    a = (a+0xd3a2646c) ^ (a<<9);
    a = (a+0xfd7046c5) + (a<<3);
    a = (a^0xb55a4f09) ^ (a>>16);
    return a;
}

//Writes a given fragment to a fragment buffer at a given location
__host__ __device__ void writeToDepthbuffer(int x, int y, fragment frag, fragment* depthbuffer, glm::vec2 resolution){
  if(x<resolution.x && y<resolution.y){
    int index = (y*resolution.x) + x;
    depthbuffer[index] = frag;
  }
}

//Reads a fragment from a given location in a fragment buffer
__host__ __device__ fragment getFromDepthbuffer(int x, int y, fragment* depthbuffer, glm::vec2 resolution){
  if(x<resolution.x && y<resolution.y){
    int index = (y*resolution.x) + x;
    return depthbuffer[index];
  }else{
    fragment f;
    return f;
  }
}

//Writes a given pixel to a pixel buffer at a given location
__host__ __device__ void writeToFramebuffer(int x, int y, glm::vec3 value, glm::vec3* framebuffer, glm::vec2 resolution){
  if(x<resolution.x && y<resolution.y){
    int index = (y*resolution.x) + x;
    framebuffer[index] = value;
  }
}

//Reads a pixel from a pixel buffer at a given location
__host__ __device__ glm::vec3 getFromFramebuffer(int x, int y, glm::vec3* framebuffer, glm::vec2 resolution){
  if(x<resolution.x && y<resolution.y){
    int index = (y*resolution.x) + x;
    return framebuffer[index];
  }else{
    return glm::vec3(0,0,0);
  }
}

//Kernel that clears a given pixel buffer with a given color
__global__ void clearImage(glm::vec2 resolution, glm::vec3* image, glm::vec3 color){
    int x = (blockIdx.x * blockDim.x) + threadIdx.x;
    int y = (blockIdx.y * blockDim.y) + threadIdx.y;
    int index = x + (y * resolution.x);
    if(x<=resolution.x && y<=resolution.y){
      image[index] = color;
    }
}

//Kernel that clears a given fragment buffer with a given fragment
__global__ void clearDepthBuffer(glm::vec2 resolution, fragment* buffer, fragment frag){
    int x = (blockIdx.x * blockDim.x) + threadIdx.x;
    int y = (blockIdx.y * blockDim.y) + threadIdx.y;
    int index = x + (y * resolution.x);
    if(x<=resolution.x && y<=resolution.y){
      fragment f = frag;
      f.position.x = x;
      f.position.y = y;
      buffer[index] = f;
    }
}

//Kernel that writes the image to the OpenGL PBO directly. 
__global__ void sendImageToPBO(uchar4* PBOpos, glm::vec2 resolution, glm::vec3* image){
  
  int x = (blockIdx.x * blockDim.x) + threadIdx.x;
  int y = (blockIdx.y * blockDim.y) + threadIdx.y;
  int index = x + (y * resolution.x);
  
  if(x<=resolution.x && y<=resolution.y){

      glm::vec3 color;      
      color.x = image[index].x*255.0;
      color.y = image[index].y*255.0;
      color.z = image[index].z*255.0;

      if(color.x>255){
        color.x = 255;
      }

      if(color.y>255){
        color.y = 255;
      }

      if(color.z>255){
        color.z = 255;
      }
      
      // Each thread writes one pixel location in the texture (textel)
      PBOpos[index].w = 0;
      PBOpos[index].x = color.x;     
      PBOpos[index].y = color.y;
      PBOpos[index].z = color.z;
  }
}

//"xyCoords" are the FLOATING-POINT, sub-pixel-accurate location to be writte to
__device__ void writePointInTriangle(triangle currTri, glm::vec2 xyCoords, fragment* depthBuffer, glm::vec2 resolution){
	fragment currFrag;
	currFrag.color = currTri.c0; //assume the tri is all one color for now.
	glm::vec3 currBaryCoords = calculateBarycentricCoordinate(currTri, xyCoords);
	float fragZ = getZAtCoordinate(currBaryCoords, currTri);
	currFrag.position = glm::vec3(xyCoords.x, xyCoords.y, fragZ);
	int pixX = roundf(xyCoords.x);
	int pixY = roundf(xyCoords.y);
	//TODO: incorporate the normal in here **somewhere**
	writeToDepthbuffer(pixX, pixY, currFrag, depthBuffer, resolution);
}

//Based on slide 75-76 of the CIS560 notes, Norman I. Badler, University of Pennsylvania. 
//returns the number of pixels drawn
__device__ int rasterizeLine(glm::vec3 start, glm::vec3 finish, fragment* depthBuffer, glm::vec2 resolution, triangle currTri){
	float X, Y, Xinc, Yinc, LENGTH;
	Xinc = finish.x - start.x;
	Yinc = finish.y - start.y;
	int pixelsDrawn = 0;
	//if both zero, then we just draw a point.
	if( (abs(Xinc) < NATHANS_EPSILON) && (abs(Yinc) < NATHANS_EPSILON) ){
		writePointInTriangle(currTri, glm::vec2(start.x, start.y), depthBuffer, resolution);
		pixelsDrawn++;
	} else { //this is a line segment
		//LENGTH is the greater of Xinc, Yinc
		if(abs(Yinc) > abs(Xinc)){
			LENGTH = abs(Yinc);
			Xinc = Xinc / LENGTH; //note float division
			Yinc = 1.0; //step along Y by pixels
		} else {
			LENGTH = abs(Xinc);
			Yinc = Yinc / LENGTH; //note float division
			Xinc = 1.0; //step along X by pixels
		}
		X = start.x;
		Y = start.y;
		for(int i = 0; i <= roundf(LENGTH); i++){ //do this at least once
			writePointInTriangle(currTri, glm::vec2(X, Y), depthBuffer, resolution);
			pixelsDrawn++;
			X += Xinc;
			Y += Yinc;
		}
	} //end else 'this is a line segment'
	return pixelsDrawn;
}

__global__ void vertexShadeKernel(float* vbo, int vbosize, glm::mat4 cameraMat, glm::vec2 resolution){
  int index = (blockIdx.x * blockDim.x) + threadIdx.x;
  if(index<vbosize/3){ //each thread acts per vertex.
	  int vertNum = 3*index;
	  glm::vec4 currVert(vbo[vertNum], vbo[vertNum+1], vbo[vertNum+2], 1);
	  glm::vec4 projectedVert = cameraMat * currVert;
	  float xWinNDC = (projectedVert.x + 1)/2.0f; //shift to window NDC space (between 0 and 1)
	  float yWinNDC = (projectedVert.y + 1)/2.0f; //shift to window NDC space (between 0 and 1)
	  vbo[vertNum] = xWinNDC * resolution.x;
	  vbo[vertNum+1] = yWinNDC * resolution.y;
	  vbo[vertNum+2] = projectedVert.z; //no need to change this when shifting to window NDC space
  }
}

__global__ void primitiveAssemblyKernel(float* vbo, int vbosize, float* cbo, int cbosize, int* ibo, int ibosize, triangle* primitives){
  int index = (blockIdx.x * blockDim.x) + threadIdx.x;
  int primitivesCount = ibosize/3;
  if(index<primitivesCount){ //one thread per primitive
	  int primNum = 3*index;
	  triangle currTri;
	  int ind0 = ibo[primNum];
	  currTri.p0 = glm::vec3(vbo[3*ind0], vbo[3*ind0 + 1], vbo[3*ind0 + 2]);
	  currTri.c0 = glm::vec3(cbo[3*ind0], cbo[3*ind0 + 1], cbo[3*ind0 + 2]);
	  int ind1 = ibo[primNum + 1];
	  currTri.p1 = glm::vec3(vbo[3*ind1], vbo[3*ind1 + 1], vbo[3*ind1 + 2]);
	  currTri.c1 = glm::vec3(cbo[3*ind1], cbo[3*ind1 + 1], cbo[3*ind1 + 2]);
	  int ind2 = ibo[primNum + 2];
	  currTri.p2 = glm::vec3(vbo[3*ind2], vbo[3*ind2 + 1], vbo[3*ind2 + 2]);
	  currTri.c2 = glm::vec3(cbo[3*ind2], cbo[3*ind2 + 1], cbo[3*ind2 + 2]);
	  primitives[index] = currTri;
  }
}

//TODO: Implement a rasterization method, such as scanline.
//NATHAN: at each fragment, calculate the barycentric coordinates, and interpolate position/color. 
//for now the normal can just be the cross product of the vectors that make up the face (flat shading).
//NATHAN: add early-z here.
__global__ void rasterizationKernel(triangle* primitives, int primitivesCount, fragment* depthbuffer, glm::vec2 resolution){
  int index = (blockIdx.x * blockDim.x) + threadIdx.x;
  if(index<primitivesCount){
	  //first rasterize the OUTLINES of the triangle

	  //use recursive flood fill starting at the CENTER of the triangle (interpolate using barycentric, map back to screen space)
	  //take pixels, map them back to NDC, test to see if inside triangle (using barycentric)
	  //i think the real speedup comes from backface culling - don't rasterize the triangle at all if the winding order is "wrong"
  }
}

//TODO: Implement a fragment shader
__global__ void fragmentShadeKernel(fragment* depthbuffer, glm::vec2 resolution){
  int x = (blockIdx.x * blockDim.x) + threadIdx.x;
  int y = (blockIdx.y * blockDim.y) + threadIdx.y;
  int index = x + (y * resolution.x);
  if(x<=resolution.x && y<=resolution.y){
  }
}

//Writes fragment colors to the framebuffer
__global__ void render(glm::vec2 resolution, fragment* depthbuffer, glm::vec3* framebuffer){

  int x = (blockIdx.x * blockDim.x) + threadIdx.x;
  int y = (blockIdx.y * blockDim.y) + threadIdx.y;
  int index = x + (y * resolution.x);

  if(x<=resolution.x && y<=resolution.y){
    framebuffer[index] = depthbuffer[index].color;
  }
}

// Wrapper for the __global__ call that sets up the kernel calls and does a ton of memory management
void cudaRasterizeCore(uchar4* PBOpos, glm::vec2 resolution, float frame, float* vbo, int vbosize, float* cbo, int cbosize, int* ibo, int ibosize){

  // set up crucial magic
  int tileSize = 8;
  dim3 threadsPerBlock(tileSize, tileSize);
  dim3 fullBlocksPerGrid((int)ceil(float(resolution.x)/float(tileSize)), (int)ceil(float(resolution.y)/float(tileSize)));

  //set up framebuffer
  framebuffer = NULL;
  hipMalloc((void**)&framebuffer, (int)resolution.x*(int)resolution.y*sizeof(glm::vec3));
  
  //set up depthbuffer
  depthbuffer = NULL;
  hipMalloc((void**)&depthbuffer, (int)resolution.x*(int)resolution.y*sizeof(fragment));

  //kernel launches to black out accumulated/unaccumlated pixel buffers and clear our scattering states
  clearImage<<<fullBlocksPerGrid, threadsPerBlock>>>(resolution, framebuffer, glm::vec3(0,0,0));
  
  fragment frag;
  frag.color = glm::vec3(0,0,0);
  frag.normal = glm::vec3(0,0,0);
  frag.position = glm::vec3(0,0,-10000);
  clearDepthBuffer<<<fullBlocksPerGrid, threadsPerBlock>>>(resolution, depthbuffer,frag);

  //------------------------------
  //memory stuff
  //------------------------------
  primitives = NULL;
  hipMalloc((void**)&primitives, (ibosize/3)*sizeof(triangle));

  device_ibo = NULL;
  hipMalloc((void**)&device_ibo, ibosize*sizeof(int));
  hipMemcpy( device_ibo, ibo, ibosize*sizeof(int), hipMemcpyHostToDevice);

  device_vbo = NULL;
  hipMalloc((void**)&device_vbo, vbosize*sizeof(float));
  hipMemcpy( device_vbo, vbo, vbosize*sizeof(float), hipMemcpyHostToDevice);

  device_cbo = NULL;
  hipMalloc((void**)&device_cbo, cbosize*sizeof(float));
  hipMemcpy( device_cbo, cbo, cbosize*sizeof(float), hipMemcpyHostToDevice);

  tileSize = 32;
  int primitiveBlocks = ceil(((float)vbosize/3)/((float)tileSize));

  //------------------------------
  //vertex shader
  //------------------------------
  //hardcoding the camera for now:
  float fovy = 45.0f;
  float zNear = 0.1f;
  float zFar = 100.0f;
  float aspectRatio = resolution.x / resolution.y;
  glm::vec3 up(0,1,0);
  glm::vec3 center(0,0,0);
  glm::vec3 eye(0,0,1);
  glm::mat4 projection = glm::perspective(fovy, aspectRatio, zNear, zFar);
  glm::mat4 view = glm::lookAt(eye, center, up);
  glm::mat4 cameraMat = projection*view;
  vertexShadeKernel<<<primitiveBlocks, tileSize>>>(device_vbo, vbosize, cameraMat, resolution);
  float* transformedVerts = new float[vbosize];
  hipMemcpy( transformedVerts, device_vbo, vbosize*sizeof(float), hipMemcpyDeviceToHost);
  delete transformedVerts;

  hipDeviceSynchronize();
  //------------------------------
  //primitive assembly
  //------------------------------
  primitiveBlocks = ceil(((float)ibosize/3)/((float)tileSize));
  primitiveAssemblyKernel<<<primitiveBlocks, tileSize>>>(device_vbo, vbosize, device_cbo, cbosize, device_ibo, ibosize, primitives);
  triangle* assembledTris = new triangle[ibosize/3];
  hipMemcpy( assembledTris, primitives, (ibosize/3)*sizeof(triangle), hipMemcpyDeviceToHost);
  delete assembledTris;

  hipDeviceSynchronize();
  //------------------------------
  //rasterization
  //------------------------------
  //first draw the outlines of the triangle
  //drawOutlinesKernel<<<primitiveBlocks, tileSize>>>(primitives, ibosize/3, depthbuffer, resolution);

  rasterizationKernel<<<primitiveBlocks, tileSize>>>(primitives, ibosize/3, depthbuffer, resolution);

  hipDeviceSynchronize();
  //------------------------------
  //fragment shader
  //------------------------------
  fragmentShadeKernel<<<fullBlocksPerGrid, threadsPerBlock>>>(depthbuffer, resolution);

  hipDeviceSynchronize();
  //------------------------------
  //write fragments to framebuffer
  //------------------------------
  render<<<fullBlocksPerGrid, threadsPerBlock>>>(resolution, depthbuffer, framebuffer);
  sendImageToPBO<<<fullBlocksPerGrid, threadsPerBlock>>>(PBOpos, resolution, framebuffer);

  hipDeviceSynchronize();

  kernelCleanup();

  checkCUDAError("Kernel failed!");
}

void kernelCleanup(){
  hipFree( primitives );
  hipFree( device_vbo );
  hipFree( device_cbo );
  hipFree( device_ibo );
  hipFree( framebuffer );
  hipFree( depthbuffer );
}

