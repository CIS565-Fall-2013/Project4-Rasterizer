#include "hip/hip_runtime.h"
// CIS565 CUDA Rasterizer: A simple rasterization pipeline for Patrick Cozzi's CIS565: GPU Computing at the University of Pennsylvania
// Written by Yining Karl Li, Copyright (c) 2012 University of Pennsylvania

#include <stdio.h>
#include <hip/hip_runtime.h>
#include <cmath>
#include <thrust/random.h>
#include "rasterizeKernels.h"
#include "rasterizeTools.h"

#if CUDA_VERSION >= 5000
    #include <hip/hip_vector_types.h>
#else
    #include <cutil_math.h>
#endif

glm::vec3* framebuffer;
fragment* depthbuffer; 
float* device_vbo;
float* copy_vbo;
float* device_cbo;
float* device_nbo;
int* device_ibo;
triangle* primitives;


void checkCUDAError(const char *msg) {
  hipError_t err = hipGetLastError();
  if( hipSuccess != err) {
    fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString( err) ); 
    exit(EXIT_FAILURE); 
  }
} 

//Handy dandy little hashing function that provides seeds for random number generation
__host__ __device__ unsigned int hash(unsigned int a){
    a = (a+0x7ed55d16) + (a<<12);
    a = (a^0xc761c23c) ^ (a>>19);
    a = (a+0x165667b1) + (a<<5);
    a = (a+0xd3a2646c) ^ (a<<9);
    a = (a+0xfd7046c5) + (a<<3);
    a = (a^0xb55a4f09) ^ (a>>16);
    return a;
}

//Writes a given fragment to a fragment buffer at a given location
__host__ __device__ void writeToDepthbuffer(int x, int y, fragment frag, fragment* depthbuffer, glm::vec2 resolution){
  if(x<resolution.x && y<resolution.y){
    int index = (y*resolution.x) + x;
    depthbuffer[index] = frag;
  }
}

//Reads a fragment from a given location in a fragment buffer
__host__ __device__ fragment getFromDepthbuffer(int x, int y, fragment* depthbuffer, glm::vec2 resolution){
  if(x<resolution.x && y<resolution.y){
    int index = (y*resolution.x) + x;
    return depthbuffer[index];
  }else{
    fragment f;
    return f;
  }
}

//Writes a given pixel to a pixel buffer at a given location
__host__ __device__ void writeToFramebuffer(int x, int y, glm::vec3 value, glm::vec3* framebuffer, glm::vec2 resolution){
  if(x<resolution.x && y<resolution.y){
    int index = (y*resolution.x) + x;
    framebuffer[index] = value;
  }
}

//Reads a pixel from a pixel buffer at a given location
__host__ __device__ glm::vec3 getFromFramebuffer(int x, int y, glm::vec3* framebuffer, glm::vec2 resolution){
  if(x<resolution.x && y<resolution.y){
    int index = (y*resolution.x) + x;
    return framebuffer[index];
  }else{
    return glm::vec3(0,0,0);
  }
}

//Kernel that clears a given pixel buffer with a given color
__global__ void clearImage(glm::vec2 resolution, glm::vec3* image, glm::vec3 color){
    int x = (blockIdx.x * blockDim.x) + threadIdx.x;
    int y = (blockIdx.y * blockDim.y) + threadIdx.y;
    int index = x + (y * resolution.x);
    if(x<=resolution.x && y<=resolution.y){
      image[index] = color;
    }
}

//Kernel that clears a given fragment buffer with a given fragment
__global__ void clearDepthBuffer(glm::vec2 resolution, fragment* buffer, fragment frag){
    int x = (blockIdx.x * blockDim.x) + threadIdx.x;
    int y = (blockIdx.y * blockDim.y) + threadIdx.y;
    int index = x + (y * resolution.x);
    if(x<=resolution.x && y<=resolution.y){
      fragment f = frag;
      f.position.x = x;
      f.position.y = y;
      buffer[index] = f;
    }
}

//Kernel that writes the image to the OpenGL PBO directly. 
__global__ void sendImageToPBO(uchar4* PBOpos, glm::vec2 resolution, glm::vec3* image){
  
  int x = (blockIdx.x * blockDim.x) + threadIdx.x;
  int y = (blockIdx.y * blockDim.y) + threadIdx.y;
  int index = x + (y * resolution.x);
  
  if(x<=resolution.x && y<=resolution.y){

      glm::vec3 color;      
      color.x = image[index].x*255.0;
      color.y = image[index].y*255.0;
      color.z = image[index].z*255.0;

      if(color.x>255){
        color.x = 255;
      }

      if(color.y>255){
        color.y = 255;
      }

      if(color.z>255){
        color.z = 255;
      }
      
      // Each thread writes one pixel location in the texture (textel)
      PBOpos[index].w = 0;
      PBOpos[index].x = color.x;     
      PBOpos[index].y = color.y;
      PBOpos[index].z = color.z;
  }
}
__host__ __device__ void printvec3(glm::vec3 vec)
{
	printf("b %f,%f,%f \n",vec.x,vec.y,vec.z);
}
__host__ __device__ void printfloat3(float x, float y, float z)
{
	printf("point %f,%f,%f \n",x,y,z);
}

//TODO: Implement a vertex shader
__global__ void vertexShadeKernel(float* vbo, int vbosize
	,float* nbo // transformation
	,glm::mat4 modelM, glm::mat4 viewM, glm::mat4 projectionM
	)
{
  int index = (blockIdx.x * blockDim.x) + threadIdx.x;
  if(index<vbosize/3){
	  glm::mat4 mvp = projectionM * viewM * modelM; // 	  
	  glm::vec4 vertex(vbo[index*3],vbo[index*3+1],vbo[index*3+2],1.0);
	  glm::vec4 newVertex = mvp * vertex; //vertex in clip coordinate	
	  
	  vbo[index*3] = newVertex.x; 
	  vbo[index*3+1] = newVertex.y; 
	  vbo[index*3+2] = newVertex.z;	  

	  glm::vec4 newNormal = modelM * glm::vec4(nbo[index*3],nbo[index*3+1],nbo[index*3+2],1.0);
	  glm::vec3 normal = glm::normalize(glm::vec3(newNormal));	  
	  nbo[index*3] = normal.x; 
	  nbo[index*3+1] = normal.y;
	  nbo[index*3+2] = normal.z;  

  }
}

//TODO: Implement primative assembly
__global__ void primitiveAssemblyKernel(float* vbo,int vbosize, float* cbo, int cbosize, int* ibo, int ibosize, 
	float* nbo, int nbosize,
	float* copyvbo, glm::mat4 modelM,
	glm::vec3 camPos,
	light lit,
	triangle* primitives)
{
  int index = (blockIdx.x * blockDim.x) + threadIdx.x;
  int primitivesCount = ibosize/3;
  if(index<primitivesCount){	
	  

	  primitives[index].p0 = glm::vec3(vbo[ibo[index*3]*3],vbo[ibo[index*3]*3+1],vbo[ibo[index*3]*3+2]);
	  primitives[index].p1 = glm::vec3(vbo[ibo[index*3+1]*3],vbo[ibo[index*3+1]*3+1],vbo[ibo[index*3+1]*3+2]);
	  primitives[index].p2 = glm::vec3(vbo[ibo[index*3+2]*3],vbo[ibo[index*3+2]*3+1],vbo[ibo[index*3+2]*3+2]);

	  //get vertex position in world space
	  glm::vec4 vertexWorldP0(copyvbo[ibo[index*3]*3],copyvbo[ibo[index*3]*3+1],copyvbo[ibo[index*3]*3+2],1.0);
	  glm::vec4 vertexWorldP1(copyvbo[ibo[index*3+1]*3],copyvbo[ibo[index*3+1]*3+1],copyvbo[ibo[index*3+1]*3+2],1.0);
	  glm::vec4 vertexWorldP2(copyvbo[ibo[index*3+2]*3],copyvbo[ibo[index*3+2]*3+1],copyvbo[ibo[index*3+2]*3+2],1.0);
	  vertexWorldP0 = modelM * vertexWorldP0;
	  vertexWorldP1 = modelM * vertexWorldP1;
	  vertexWorldP2 = modelM * vertexWorldP2;
	  primitives[index].wp0 = glm::vec3(vertexWorldP0.x,vertexWorldP0.y,vertexWorldP0.z);
	  primitives[index].wp1 = glm::vec3(vertexWorldP1.x,vertexWorldP1.y,vertexWorldP1.z);
	  primitives[index].wp2 = glm::vec3(vertexWorldP2.x,vertexWorldP2.y,vertexWorldP2.z);


	/*  if(cbosize == vbosize)
	  {
			primitives[index].c0 = glm::vec3(cbo[ibo[index*3]*3],cbo[ibo[index*3]*3+1],cbo[ibo[index*3]*3+2]);
			primitives[index].c1 = glm::vec3(cbo[ibo[index*3+1]*3],cbo[ibo[index*3+1]*3+1],cbo[ibo[index*3+1]*3+2]);
			primitives[index].c2 = glm::vec3(cbo[ibo[index*3+2]*3],cbo[ibo[index*3+2]*3+1],cbo[ibo[index*3+2]*3+2]);
	  }
	  else
	  {
			primitives[index].c0 = glm::vec3(cbo[0],cbo[1],cbo[2]);
			primitives[index].c1 = glm::vec3(cbo[3],cbo[4],cbo[5]);
			primitives[index].c2 = glm::vec3(cbo[6],cbo[7],cbo[8]);
	  }*/
	  primitives[index].c0 = lit.bgColor;
	  primitives[index].c1 = lit.bgColor;
	  primitives[index].c2 = lit.bgColor;
	  //normal
	  primitives[index].n0 = glm::vec3(nbo[ibo[index*3]*3],nbo[ibo[index*3]*3+1],nbo[ibo[index*3]*3+2]);
	  primitives[index].n1 = glm::vec3(nbo[ibo[index*3+1]*3],nbo[ibo[index*3+1]*3+1],nbo[ibo[index*3+1]*3+2]);
	  primitives[index].n2 = glm::vec3(nbo[ibo[index*3+2]*3],nbo[ibo[index*3+2]*3+1],nbo[ibo[index*3+2]*3+2]);
	  
#if BACKFACECULL == 1	 
	  //back face culling
	  if(glm::dot(camPos - primitives[index].wp0,primitives[index].n0)<0 
		  && glm::dot(camPos - primitives[index].wp1,primitives[index].n1)<0
		  && glm::dot(camPos - primitives[index].wp2,primitives[index].n2)<0
		  )
	  {
		 // printf("backface");
		  primitives[index].isCulled = true;
	  }
	  else
	  {
		  primitives[index].isCulled = false;
	  }
#endif
  }
}

//TODO: Implement a rasterization method, such as scanline.
__global__ void rasterizationKernel(triangle* primitives, int primitivesCount, fragment* depthbuffer, glm::vec2 resolution,material mat,light lit){
  int index = (blockIdx.x * blockDim.x) + threadIdx.x;
  if(index<primitivesCount){

#if BACKFACECULL == 1	 
	  if(primitives[index].isCulled)
		  return;
#endif
	  //get triangle bounding box
	  glm::vec3 boxMin(0);
	  glm::vec3 boxMax(0);
	  getAABBForTriangle(primitives[index],boxMin,boxMax);
	  /*printvec3(boxMin);
	  printvec3(boxMax);*/

	  //get corresponding pixel for bounding box
	  glm::vec2 pixelMin = convertWorldToPixel(boxMin,resolution);
	  glm::vec2 pixelMax = convertWorldToPixel(boxMax,resolution);


	  pixelMin.x = max((int)pixelMin.x,0);
	  pixelMax.x = min((int)pixelMax.x,(int)resolution.x-1);
	  pixelMax.y = max((int)pixelMax.y,0);
	  pixelMin.y = min((int)pixelMin.y,(int)resolution.y-1);

	  //printf("pixels: %f,%f,%f,%f",pixelMin.x,pixelMin.y,pixelMax.x,pixelMax.y);
	  //loop from ymin to ymax
	
	  fragment tmpFrag;
	  int fragIdx = 0;
	  for(int y = pixelMax.y; y <= pixelMin.y; y++)
	  {
		 
		  //loop from xmin to xmax
		  for(int x = pixelMin.x; x <= pixelMax.x; x++)
		  {
			  fragIdx = x + y * resolution.x;
			 
			  //get pixel position in Canonical View Volumes
			  glm::vec2 pixelPoint;
			  pixelPoint.x = (2.0 * x / (float)resolution.x) - 1;
			  pixelPoint.y = 1-(2.0 * y / (float)resolution.y);

			  //get barycentricCoordinate
			  glm::vec3 barycCoord = calculateBarycentricCoordinate(primitives[index],pixelPoint);
			  //check if pixel is within the triangle
			  if(!isBarycentricCoordInBounds(barycCoord))
			  {
				  continue;
			  }

			  //get depth value
			  float depth = getZAtCoordinate(barycCoord,primitives[index]);
			 
			  //in normalized device coordinate
			  tmpFrag.position = glm::vec3(pixelPoint.x,pixelPoint.y,depth);
			  //color interpolation
			  //tmpFrag.color = barycCoord.x * primitives[index].c0 + barycCoord.y * primitives[index].c1 + barycCoord.z * primitives[index].c2;			  			
			  tmpFrag.color = mat.diffuseColor;
			  tmpFrag.normal = (primitives[index].n0 + primitives[index].n1 + primitives[index].n2)/(float)3.0;
			  tmpFrag.hasMatColor = 1; // has material color
			  bool wait = true;
			  while(wait)
			  {
				  if(atomicExch(&(depthbuffer[fragIdx].isLock),1) == 0)
				  {
					  if(depthbuffer[fragIdx].position.x <= -10000 || tmpFrag.position.z > depthbuffer[fragIdx].position.z)
					  {
						  depthbuffer[fragIdx] = tmpFrag;							 
					  }
					  depthbuffer[fragIdx].isLock = 0;
					  wait = false;
				  }
			  }
		  
		  }//end loop for x
	  }//end loop for y

  }
}

//TODO: Implement a fragment shader
__global__ void fragmentShadeKernel(fragment* depthbuffer, glm::vec2 resolution,glm::vec3 cameraPos
	,light lit
	,material mat)
{
  int x = (blockIdx.x * blockDim.x) + threadIdx.x;
  int y = (blockIdx.y * blockDim.y) + threadIdx.y;
  int index = x + (y * resolution.x);
  if(x<=resolution.x && y<=resolution.y){
	  
	  if(depthbuffer[index].normal == glm::vec3(0,0,0))
	  {
		  //printf("bg area");
		  return;
	  }
#if PHONE == 1
	  //phone 
	  glm::vec3 point = depthbuffer[index].position;
	  glm::vec3 normal = depthbuffer[index].normal;
	  glm::vec3 Lvector = point - lit.pos;
	  Lvector = glm::normalize(Lvector);
	  float diffuse = glm::dot(normal,Lvector);
	  diffuse = max(diffuse,0.0f);
	  glm::vec3 diffuseColor = diffuse * depthbuffer[index].color;

	  
	  glm::vec3 reVector = Lvector - 2.0f * glm::dot(Lvector,normal);
	  glm::vec3 eyeToPoint = cameraPos - point;
	  eyeToPoint = glm::normalize(eyeToPoint);
	  float specular = pow(glm::dot(eyeToPoint,reVector),10.0f);
	  specular = max(specular,0.0f);
	  glm::vec3 specularColor = mat.specularColor * specular;
	  depthbuffer[index].color = lit.emitPower * lit.color * (mat.kd * diffuseColor + mat.ks * specularColor) + mat.ka * lit.ambientLColor;
#elif DEPTH == 1
	//depth buffer rendering
	depthbuffer[index].color = glm::vec3(1-((abs(depthbuffer[index].position.z)-18)/3.5)); 
	depthbuffer[index].color = glm::clamp(depthbuffer[index].color,glm::vec3(0,0,0),glm::vec3(1,1,1));
#endif
  }
}

//Writes fragment colors to the framebuffer
__global__ void render(glm::vec2 resolution, fragment* depthbuffer, glm::vec3* framebuffer){

  int x = (blockIdx.x * blockDim.x) + threadIdx.x;
  int y = (blockIdx.y * blockDim.y) + threadIdx.y;
  int index = x + (y * resolution.x);

  if(x<=resolution.x && y<=resolution.y){

    framebuffer[index] = depthbuffer[index].color;
#if ANTIA == 1
	if(depthbuffer[index].hasMatColor != 0)
	{
		int count = 0;
		glm::vec3 color(0,0,0);
		for(int i = -1;i<=1;i++)
		{
			for(int j = -1;j<=1;j++)
			{
				if(x+i >=0 && x+i < resolution.x && y+j >=0 && y+j <resolution.y)
				{
					int idx = (x+i)+(y+j)*resolution.x;
					color += depthbuffer[idx].color;
					count ++;
				}						
			}
		}
		color /= (float)count;
		framebuffer[index] = color;
	}
	
#endif
  }
}

// Wrapper for the __global__ call that sets up the kernel calls and does a ton of memory management
void cudaRasterizeCore(uchar4* PBOpos, glm::vec2 resolution, float frame, float* vbo, int vbosize, float* cbo, int cbosize, int* ibo, int ibosize,
	float* nbo, int nbosize
	,glm::mat4 modelM, glm::mat4 viewM, glm::mat4 projectionM
	,glm::vec3* images
	,glm::vec3 cameraPos
	)
{

  // set up crucial magic
  int tileSize = 8;
  dim3 threadsPerBlock(tileSize, tileSize);
  dim3 fullBlocksPerGrid((int)ceil(float(resolution.x)/float(tileSize)), (int)ceil(float(resolution.y)/float(tileSize)));

  

  //set up framebuffer
  framebuffer = NULL;
  hipMalloc((void**)&framebuffer, (int)resolution.x*(int)resolution.y*sizeof(glm::vec3));

  //set up depthbuffer
  depthbuffer = NULL;
  hipMalloc((void**)&depthbuffer, (int)resolution.x*(int)resolution.y*sizeof(fragment));
  
  //kernel launches to black out accumulated/unaccumlated pixel buffers and clear our scattering states
  clearImage<<<fullBlocksPerGrid, threadsPerBlock>>>(resolution, framebuffer, glm::vec3(0,0,0));
  
  fragment frag;
  frag.color = glm::vec3(0,0,0);
  frag.normal = glm::vec3(0,0,0);
  frag.position = glm::vec3(0,0,-10000);
  frag.isLock = 0;
  frag.hasMatColor = 0;
  clearDepthBuffer<<<fullBlocksPerGrid, threadsPerBlock>>>(resolution, depthbuffer,frag);

  //------------------------------
  //memory stuff
  //------------------------------
  primitives = NULL;
  hipMalloc((void**)&primitives, (ibosize/3)*sizeof(triangle));

  device_ibo = NULL;
  hipMalloc((void**)&device_ibo, ibosize*sizeof(int));
  hipMemcpy( device_ibo, ibo, ibosize*sizeof(int), hipMemcpyHostToDevice);

  device_vbo = NULL;
  hipMalloc((void**)&device_vbo, vbosize*sizeof(float));
  hipMemcpy( device_vbo, vbo, vbosize*sizeof(float), hipMemcpyHostToDevice);

  //store position information before transform to clip coordinate
  copy_vbo = NULL;
  hipMalloc((void**)&copy_vbo, vbosize*sizeof(float));
  hipMemcpy( copy_vbo, vbo, vbosize*sizeof(float), hipMemcpyHostToDevice);

  device_cbo = NULL;
  hipMalloc((void**)&device_cbo, cbosize*sizeof(float));
  hipMemcpy( device_cbo, cbo, cbosize*sizeof(float), hipMemcpyHostToDevice);

  device_nbo = NULL;
  hipMalloc((void**)&device_nbo, nbosize*sizeof(float));
  hipMemcpy( device_nbo, nbo, nbosize*sizeof(float), hipMemcpyHostToDevice);


  tileSize = 32;
  int primitiveBlocks = ceil(((float)vbosize/3)/((float)tileSize));

  light lit;
  lit.color = glm::vec3(0.8,0.8,0.9);
  lit.pos = glm::vec3(10,-120.0f,-40.0f);
  lit.emitPower = 2.0f; 
  lit.ambientLColor = glm::vec3(1.0,1.0,1.0);
  lit.bgColor = glm::vec3(0,0,0);

  material mat;
  mat.diffuseColor = glm::vec3(1.0,1.0,0.0);
  mat.specularColor = glm::vec3(1.0,1.0,0.0);
  mat.specularCon = 2.0f;
  mat.kd = 0.6;
  mat.ks = 0.2;
  mat.ka = 0.2;

  //------------------------------
  //vertex shader
  //------------------------------
  vertexShadeKernel<<<primitiveBlocks, tileSize>>>(device_vbo, vbosize,device_nbo,modelM,viewM,projectionM); 

  hipDeviceSynchronize();
  //------------------------------
  //primitive assembly
  //------------------------------
  primitiveBlocks = ceil(((float)ibosize/3)/((float)tileSize));
  primitiveAssemblyKernel<<<primitiveBlocks, tileSize>>>(device_vbo, vbosize, device_cbo, cbosize, device_ibo, ibosize,device_nbo,nbosize,
	  copy_vbo,modelM,
	  cameraPos,
	  lit,
	  primitives);

  hipDeviceSynchronize();
  //------------------------------
  //rasterization
  //------------------------------
  rasterizationKernel<<<primitiveBlocks, tileSize>>>(primitives, ibosize/3, depthbuffer, resolution,mat,lit);

  hipDeviceSynchronize();
  //------------------------------
  //fragment shader
  //------------------------------


  fragmentShadeKernel<<<fullBlocksPerGrid, threadsPerBlock>>>(depthbuffer, resolution,cameraPos,lit,mat);

  hipDeviceSynchronize();
  //------------------------------
  //write fragments to framebuffer
  //------------------------------
  render<<<fullBlocksPerGrid, threadsPerBlock>>>(resolution, depthbuffer, framebuffer);
  sendImageToPBO<<<fullBlocksPerGrid, threadsPerBlock>>>(PBOpos, resolution, framebuffer);

  //retrieve image from GPU
  hipMemcpy(images, framebuffer,(int)resolution.x*(int)resolution.y*sizeof(glm::vec3), hipMemcpyDeviceToHost);

  hipDeviceSynchronize();

  kernelCleanup();

  checkCUDAError("Kernel failed!");
}

void kernelCleanup(){
  hipFree( primitives );
  hipFree( device_vbo );
  hipFree( device_cbo );
  hipFree( device_ibo );
  hipFree( framebuffer );
  hipFree( depthbuffer );
  hipFree( copy_vbo);
  hipFree( device_nbo );
}

